#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define THREADS_PER_BLOCK_LIMIT 1024
#define BLOCKS_PER_GRID_LIMIT 65535

int threadsPerBlock = 256;
int maxBlocksPerGrid = 4096;

__global__ void device_scalar_matrix_mult(int datasetSize, float scalar, float *matrixDeviceRows)
{
    int numThreads = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int positionsToMultiply = (datasetSize + (numThreads - 1)) / numThreads;
    int initialPos = index * positionsToMultiply;
    int currIndex = initialPos;
    
    float *auxMatrixPtr = matrixDeviceRows + initialPos;

    if(initialPos >= datasetSize)
    {
        return;
    }

    for (int i = 0 ; i < positionsToMultiply ; i++, auxMatrixPtr++)
    {
        if (initialPos + i < datasetSize)
        {
            *auxMatrixPtr = scalar * *auxMatrixPtr;
        }
    }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix)
{
    int datasetSize = matrix->height * matrix->width;

    hipError_t hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, datasetSize * sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (matrix->h_rows -> matrix->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    device_scalar_matrix_mult<<<maxBlocksPerGrid, threadsPerBlock>>>(datasetSize, scalar_value, matrix->d_rows);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, datasetSize * sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (matrix->d_rows -> matrix->h_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    return 1;
}

__global__ void device_matrix_matrix_mult(int datasetSize, int aWidth, float *aRows, int bWidth, float *bRows, int cWidth, float *cRows)
{
    int numThreads = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int rowsToMultiply = (datasetSize + (numThreads - 1)) / numThreads;
    int initialPos = index * rowsToMultiply;

    float *auxMatrixAPtr = aRows + initialPos * aWidth;
    float *auxMatrixBPtr = bRows;
    float *auxMatrixCPtr = cRows + initialPos * cWidth;

    int aColumn = 0;

    if(initialPos >= datasetSize)
    {
        return;
    }

    for (int aCurrRow = 0 ; aCurrRow < rowsToMultiply ; auxMatrixAPtr++)
    {
        if(initialPos + aCurrRow >= datasetSize)
        {
            break;
        }

        auxMatrixCPtr = cRows + initialPos * cWidth;
        auxMatrixCPtr += aCurrRow * cWidth;

        auxMatrixBPtr = bRows;
        auxMatrixBPtr += aColumn * bWidth;

        for (int column = 0; column < bWidth; auxMatrixBPtr++, auxMatrixCPtr++, column++)
        {
            *auxMatrixCPtr += *auxMatrixAPtr * (*auxMatrixBPtr);
        }

        if (aColumn + 1 == aWidth)
        {
            aCurrRow++;
            aColumn = 0;
        }
        else
        {
            aColumn++;
        }
    }
}

int matrix_matrix_mult(struct matrix *a, struct matrix *b, struct matrix *c)
{
    int datasetSizeA = a->height * a->width * sizeof(float);
    int datasetSizeB = b->height * b->width * sizeof(float);
    int datasetSizeC = c->height * c->width * sizeof(float);

    hipError_t hipError_t = hipMemcpy(a->d_rows, a->h_rows, datasetSizeA, hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (a->h_rows -> a->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    hipError_t = hipMemcpy(b->d_rows, b->h_rows, datasetSizeB, hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (b->h_rows -> b->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    hipError_t = hipMemcpy(c->d_rows, c->h_rows, datasetSizeC, hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (c->h_rows -> c->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    device_matrix_matrix_mult<<<maxBlocksPerGrid, threadsPerBlock>>>(c->height, a->width, a->d_rows, b->width, b->d_rows, c->width, c->d_rows);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(c->h_rows, c->d_rows, datasetSizeC, hipMemcpyDeviceToHost);
    
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (c->d_rows -> matrix->h_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    return 1;
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
    if (threads_per_block > THREADS_PER_BLOCK_LIMIT || max_blocks_per_grid > BLOCKS_PER_GRID_LIMIT || threads_per_block <= 0 || max_blocks_per_grid <= 0)
    {
        printf("ERROR! Invalid CUDA amount for blocks (%d) or threads (%d)\n", threads_per_block, max_blocks_per_grid);
        return 0;
    }

    threadsPerBlock = threads_per_block;
    maxBlocksPerGrid = max_blocks_per_grid;

    return 1;
}