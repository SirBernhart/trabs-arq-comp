#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define THREADS_PER_BLOCK_LIMIT 1024
#define BLOCKS_PER_GRID_LIMIT 65535

int threadsPerBlock = 256;
int maxBlocksPerGrid = 4096;

__global__ device_scalar_matrix_mult(int datasetSize, float scalar, float *matrixDeviceRows)
{
    int numThreads = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int positionsToMultiply = (datasetSize + (numThreads - 1)) / numThreads;
    int initialPos = index * positionsToMultiply;

    float *auxMatrixPtr = matrixDeviceRows + initialPos;

    for (int i = 0; i < positionsToMultiply; i++, auxMatrixPtr++)
    {
        if (initialPos + i < datasetSize)
        {
            auxMatrixPtr *= scalar;
        }
    }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix)
{
    int datasetSize = matrix->height * matrix->width;

    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows, datasetSize * sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (matrix->h_rows -> matrix->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    device_scalar_matrix_mult<<<maxBlocksPerGrid, threadsPerBlock>>>(datasetSize, matrix.d_rows);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrix->h_rows, matrix->d_rows, datasetSize * sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (matrix->d_rows -> matrix->h_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    return 1;
}

__global__ device_matrix_matrix_mult(int datasetSize, float *matrixADeviceRows, float *matrixBDeviceRows, float *matrixCDeviceRows)
{
    int numThreads = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int positionsToMultiply = (datasetSize + (numThreads - 1)) / numThreads;
    int initialPos = index * positionsToMultiply;

    float *auxMatrixAPtr = matrixADeviceRows + initialPos;
    float *auxMatrixBPtr = matrixBDeviceRows + initialPos;
    float *auxMatrixCPtr = matrixCDeviceRows + initialPos;

    int aColumn = 0;
    for (int aRow = 0; aRow < args->amountOfRowsToOperate; auxMatrixAPointer++)
    {
        auxMatrixCPointer = args->matCStartingRowPtr;
        auxMatrixCPointer += aRow * args->matCWidth;

        auxMatrixBPointer = args->matB->rows;
        auxMatrixBPointer += aColumn * args->matB->width;

        m256MatrixAPointer = _mm256_set1_ps(*auxMatrixAPointer);

        for (int column = 0; column < args->matB->width; auxMatrixBPointer += 8, auxMatrixCPointer += 8, column += 8)
        {
            m256MatrixBPointer = _mm256_load_ps(auxMatrixBPointer);
            m256MatrixCPointer = _mm256_load_ps(auxMatrixCPointer);
            m256MultAddResultPointer = _mm256_fmadd_ps(m256MatrixAPointer, m256MatrixBPointer, m256MatrixCPointer);

            _mm256_store_ps(auxMatrixCPointer, m256MultAddResultPointer);
        }

        if (aColumn + 1 == args->matAWidth)
        {
            aRow++;
            aColumn = 0;
        }
        else
        {
            aColumn++;
        }
    }
}

int matrix_matrix_mult(struct matrix *a, struct matrix *b, struct matrix *c)
{
    int datasetSizeA = a->height * a->width;

    int datasetSizeB = b->height * b->width;
    int datasetSizeC = c->height * c->width;

    hipError_t = hipMemcpy(a->d_rows, a->h_rows, datasetSizeA * sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (a->h_rows -> a->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }
    hipError_t = hipMemcpy(b->d_rows, b->h_rows, datasetSizeB * sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (b->h_rows -> b->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    hipError_t = hipMemcpy(c->d_rows, c->h_rows, datasetSizeC * sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (c->h_rows -> c->d_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    device_matrix_matrix_mult<<<maxBlocksPerGrid, threadsPerBlock>>>(int datasetSizeC, a->d_rows, b->d_rows, c->d_rows);

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(c->h_rows, c->d_rows, datasetSizeC * sizeof(float), hipMemcpyDeviceToHost);

    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (c->d_rows -> matrix->h_rows) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 0;
    }

    return 1;
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
    if (threads_per_block > THREADS_PER_BLOCK_LIMIT || max_blocks_per_grid > BLOCKS_PER_GRID_LIMIT || threads_per_block <= 0 || max_blocks_per_grid <= 0)
    {
        printf("ERROR! Invalid CUDA amount for blocks (%d) or threads (%d)\n", threads_per_block, max_blocks_per_grid);
        return 0;
    }

    threadsPerBlock = threads_per_block;
    maxBlocksPerGrid = max_blocks_per_grid;

    return 1;
}